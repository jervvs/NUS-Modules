#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <time.h>
#include "hash.h"

// CUDA runtime
#include <hip/hip_runtime.h>

#define DIGEST_LENGTH 52 // Size in words
#define HASH_LENGTH 32  // Size in words

int check_cuda_errors()
{
    hipError_t rc;
    rc = hipGetLastError();
    if (rc != hipSuccess)
    {
        printf("Last CUDA error %s\n", hipGetErrorString(rc));
        return 1;
    }
    return 0;
}

long long wall_clock_time()
{
#ifdef __linux__
    struct timespec tp;
    clock_gettime(CLOCK_REALTIME, &tp);
    return (long long)(tp.tv_nsec + (long long)tp.tv_sec * 1000000000ll);
#else
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (long long)(tv.tv_usec * 1000 + (long long)tv.tv_sec * 1000000000ll);
#endif
}

//TODO global hash
__global__
void globalHash(
    uint8_t *hash, 
    const uint8_t *X,  
    unsigned long long n, 
    int* found, 
    unsigned long long *res_nonce,
    unsigned long long offset
){
    //CONSTRUCTING INPUT
    uint8_t input[DIGEST_LENGTH];
    for (int i = 0; i < 44; i++) { 
        input[i] = X[i];
    }

    unsigned long long threadID = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long nonce = threadID + offset;

    for (int i = 44; i < DIGEST_LENGTH; i++) {
        input[i] = (nonce >> 8 * (DIGEST_LENGTH - i - 1)) & 0xFF;
    }

    uint8_t localHash[HASH_LENGTH];
    sha256(localHash, input, DIGEST_LENGTH);

    // Check 64-bit prefix of SHA256(digest)
    unsigned long long prefix = 0x0000;
    for (int i = 0; i < 8; i++) {
        prefix = prefix | ((unsigned long long) localHash[i] << (8*(7-i)));
    }

    if (prefix < n) {
        if (atomicExch(found, 1) == 0) {
            memcpy(hash, localHash, sizeof(uint8_t) * HASH_LENGTH);
            memcpy(res_nonce, &nonce, sizeof(unsigned long long));
        }
    }
}


int main(int argc, char *argv[]) {
    //INPUT HANDLING
    if (argc != 2 ) {
        printf("Usage: [executable] [file 1]\n");
        return 1;
    }
    
    FILE* file = fopen(argv[1], "r"); /* should check the result */
  
    char *prevDigest = (char*) malloc(64+1);
    uint8_t *tid = (uint8_t*) malloc(sizeof(uint8_t));
    unsigned long long n;

    fscanf(file, "%s", prevDigest);
    fscanf(file, "%s", tid);
    fscanf(file, "%llu", &n);
    fclose(file);

    //PREV DIGEST
    uint8_t *prev = (uint8_t*) malloc(sizeof(uint8_t) * 32);
    char *buff = prevDigest;
    for (int i = 0; i<32; i++){
        sscanf(buff, "%02hhX", &prev[i]);
        buff += 2; 
    }

    //Getting UNIX Timestamp
    uint32_t timeNow = (uint32_t) time(NULL);
    uint8_t *t = (uint8_t*) malloc(sizeof(uint8_t) * 4);
    t[0] = (uint8_t) (timeNow >> 24);
    t[1] = (uint8_t) (timeNow >> 16);
    t[2] = (uint8_t) (timeNow >> 8);
    t[3] = (uint8_t) timeNow;

    //HOST DATA
    //CHANGE GRID AND BLOCK SIZES HERE
    const uint32_t block_count = 80;
	const uint32_t threads_per_block = 256;
    const uint32_t thread_count = block_count * threads_per_block;
    
    //DEVICE DATA
    uint8_t *hash;
    uint8_t *X; 
    unsigned long long *res_nonce;
    unsigned long long offset = 0;
    int *found;  

    // "Malloc" device memory
    hipMallocManaged((void **)&hash, HASH_LENGTH * sizeof(uint8_t));
    hipMallocManaged((void **)&X, DIGEST_LENGTH * sizeof(uint8_t));
    hipMallocManaged(&res_nonce, sizeof(unsigned long long));
    hipMallocManaged(&found, sizeof(int));
    *found = 0;

    //FILL X WITH THE INPUT VALUES
    int i, j;
    for (i = 0, j = 0; i < 32; i++) { 
        X[i] = prev[j];
        j++;
    }
    for (i = 32, j = 0; i < 36; i++) { 
        X[i] = t[j];
        j++;
    }
    for (i = 36, j = 0; i < 44; i++) { 
        X[i] = tid[j];
        j++;
    }

    int start = wall_clock_time();
    while (!(*found)) {
        globalHash<<<block_count, threads_per_block>>>(hash, X, n, found, res_nonce, offset);
        hipDeviceSynchronize();
        if (check_cuda_errors()){break;}
        offset += thread_count;
    }
    int end = wall_clock_time();
    printf("The process took %1.2f seconds\n", ((float)(end - start))/1000000000);

    // OUTPUT
    printf("%d\n", timeNow);
    printf("%llu\n", *res_nonce);
    for (int i = 0; i < HASH_LENGTH; i++) {
        printf("%02x", hash[i]);
    }
    printf("\n");

    // CLEANUP
    hipFree(hash);
    hipFree(X);
    hipFree(found);
    hipFree(res_nonce);

    hipDeviceReset();
    return 0;
}